#include "hip/hip_runtime.h"
#include "rid.hpp"
#include "types.hpp"
#include "random.hpp"
#include "submatrix.hpp"
#include "handle.hpp"
#include "util.hpp"
#include "permute.hpp"
#include "timer.hpp"
#include "flops.hpp"


void RandAdapLUPP(const double *A, int m, int n, 
    int *&sk, int *&rd, double *&T, int &rank, double &flops,
    double tol, int blk) {

  Timer t;
 
  // allocate memory
  t.start();
  dvec LUmat(m*n); // same size as A
  double *LU = thrust::raw_pointer_cast(LUmat.data());
  t.stop(); double t6 = t.elapsed_time();

  // (global) permutation
  ivec P(m);
  thrust::sequence(P.begin(), P.end(), 0);

  // random Gaussian matrix
  t.start();
  dvec Gmat(n*blk);
  Random::Gaussian(Gmat, 0., 1./blk);
  double *G = thrust::raw_pointer_cast(Gmat.data());
  t.stop(); double t0 = t.elapsed_time();
  //print(G, n, blk, "G");


  // compute sample matrix
  t.start();
  auto const& handle = Handle_t::instance();
  double one = 1.0, zero = 0.;
  CHECK_CUBLAS( hipblasDgemm(handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
        m, blk, n, &one,
        A, m,
        G, n, &zero,
        LU, m) );
  t.stop(); double t1 = t.elapsed_time();
  flops = 2.*m*n*blk;
  //print(LU, m, n, "LU");
  

  int p  = std::min(m,n);
  int nb = std::ceil( p/blk );
  assert( p%blk == 0 );

  // prepare for LU factorizations
  dvec work; // working memory for LU
  ivec ipiv(blk); // local permutation
  ivec info(1); // an integer on gpu

  double t2 = 0., t4 = 0.;

  int k;
  for (int i=0; i<nb; i++) {
    k = i*blk; // number of processed rows/columns


    t.start();
    int a = m - k;
    int b = i < nb-1 ? blk : p-(nb-1)*blk;
    int lwork = 0;
    double *E = LU + k*m+k;
    CUSOLVER_CHECK( hipsolverDnDgetrf_bufferSize(handle.solver, a, b, E, m, &lwork) );

    work.resize(lwork);
    CUSOLVER_CHECK( hipsolverDnDgetrf(handle.solver, a, b, E, m, 
          thrust::raw_pointer_cast(work.data()), 
          thrust::raw_pointer_cast(ipiv.data()), 
          thrust::raw_pointer_cast(info.data()) ));

    assert( info[0]==0 );
    flops = flops + FLOPS_DGETRF(a,b);
    t.stop(); t2 += t.elapsed_time();
    //print(LU, m, n, "LU of new panel");


    // global permuation (accumulation of local permuations)
    pivots_to_permutation(ipiv, P, k);
    
    // local permutation
    ivec Phat(a);
    thrust::sequence(Phat.begin(), Phat.end(), 0);
    pivots_to_permutation(ipiv, Phat);

    //print(ipiv, "ipiv");
    //print(Phat, "Phat");
    //print(P, "P");


    if (i>0) {
      t.start();
      Permute_Matrix_Rows(Phat, LU+k, a, k, m);
      t.stop(); t4 += t.elapsed_time();
    }
    //print(LU, m, n, "LU after local permutation");


    if (i == nb-1) break;

    // next panel
    b = i < nb-2 ? blk : p-(nb-1)*blk;
    k += blk; // number of processed rows/columns
    

    // randomized sketching
    t.start();
    Random::Gaussian(Gmat, 0., 1./b);
    t.stop(); t0 += t.elapsed_time();
    //print(G, n, blk, "G");
    

    t.start();
    double *Y = LU + k*m;
    CHECK_CUBLAS( hipblasDgemm(handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
          m, b, n, &one,
          A, m,
          G, n, &zero,
          Y, m) );
    t.stop(); t1 += t.elapsed_time();
    flops = flops + 2.*m*n*b;
    //print(LU, m, n, "new sample");


    // apply global permuation
    t.start();
    Permute_Matrix_Rows(P, Y, m, b, m);
    t.stop(); t4 += t.elapsed_time();
    //print(LU, m, n, "Permute LU");


    // triangular solve
    t.start();
    CHECK_CUBLAS( hipblasDtrsm(handle.blas, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
          HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, 
          k, b, &one,
          LU, m, 
          Y, m));
    flops = flops + 1.*k*k*b;
    //print(LU, m, n, "Triangular solve");

    // Schur complement
    double negone = -1.0;
    double *S = LU + k*m+k;
    CHECK_CUBLAS( hipblasDgemm(handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
          m-k, b, k, &negone,
          LU+k, m,
          Y, m, &one,
          S, m) );
    t.stop(); t2 += t.elapsed_time();
    flops = flops + 2.*(m-k)*k*b;

    //print(LU, m, n, "Schur complement");


    // compute Frobenius norm
    auto zero = thrust::make_counting_iterator<int>(0);
    auto S_idx = thrust::make_transform_iterator(zero, SubMatrix(m-k, m));
    auto S_elm = thrust::make_permutation_iterator(dptr(S), S_idx);
    auto S_sqr = thrust::make_transform_iterator(S_elm, thrust::square<double>());
    double eSchur = thrust::reduce(S_sqr, S_sqr+(m-k)*b);

    eSchur = std::sqrt(eSchur);
    //std::cout<<"Norm of Schur complement: "<<eSchur<<std::endl;
    if (eSchur < tol) break;

  }


  CHECK_CUDA( hipMalloc((void **) &sk, sizeof(int)*k) );
  CHECK_CUDA( hipMalloc((void **) &rd, sizeof(int)*(m-k) ));
  CHECK_CUDA( hipMalloc((void **) &T,  sizeof(double)*k*(m-k) ));

  thrust::copy_n( P.begin(), k, iptr(sk) );
  thrust::copy_n( P.begin()+k, m-k, iptr(rd) );


  t.start();
  CHECK_CUBLAS( hipblasDtrsm(handle.blas, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, 
        m-k, k, &one,
        LU, m, 
        LU+k, m));

  
  auto Zero = thrust::make_counting_iterator<int>(0);
  auto indx = thrust::make_transform_iterator(Zero, SubMatrix(m-k, m));
  auto elem = thrust::make_permutation_iterator(dptr(LU+k), indx);
  thrust::copy_n( elem, k*(m-k), dptr(T) );  
  t.stop(); double t3 = t.elapsed_time();
  flops = flops + 1.*k*k*(m-k);

  
  rank = k;  // computed rank


#ifdef PROF
  std::cout<<std::endl
    <<"--------------------\n"
    <<"  RandAdapLUPP\n"
    <<"--------------------\n"
    <<"Aloc:  "<<t6<<std::endl
    <<"Rand:  "<<t0<<std::endl
    <<"GEMM:  "<<t1<<std::endl
    <<"LUPP:  "<<t2<<std::endl
    <<"Solve: "<<t3<<std::endl
    <<"Perm:  "<<t4<<std::endl
    <<"--------------------\n"
    <<"Total: "<<t0+t1+t2+t3+t4+t6<<std::endl
    <<"--------------------\n"
    <<std::endl;
#endif  
}




